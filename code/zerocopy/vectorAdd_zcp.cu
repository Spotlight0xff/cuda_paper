/*
 * Zero-Copy example, using vector addition as showcase
 */
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#define SIZE (1048576)

// CUDA kernel, using zerocopy
__global__ void
vectorAdd(float *A, float *B, float *C, int numElements)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < SIZE) {
        C[id] = A[id] + B[id];
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    float *h_A, *map_A, *h_B, *map_B, *h_C, *map_C;

    hipSetDeviceFlags(hipDeviceMapHost); // enable mapped hostmem
    hipHostAlloc(&h_A, SIZE*sizeof(float), hipHostMallocMapped);
    hipHostAlloc(&h_B, SIZE*sizeof(float), hipHostMallocMapped);
    hipHostAlloc(&h_C, SIZE*sizeof(float), hipHostMallocMapped);

    for (int i=0; i < SIZE; i++) {
        h_A[i] = rand() / (float) RAND_MAX;
        h_B[i] = rand() / (float) RAND_MAX;
    }

    hipHostGetDevicePointer(&map_A, h_A, 0); // get mapped ptr
    hipHostGetDevicePointer(&map_B, h_B, 0); // get mapped ptr
    hipHostGetDevicePointer(&map_C, h_C, 0); // get mapped ptr
    printf("> run vectorAdd using mapped host memory...\n");
    dim3 block(256);
    dim3 grid((unsigned int) ceil(SIZE / block.x));

    vectorAdd<<<grid,block>>>(h_A, h_B, h_C, SIZE);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("err: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    printf("> releasing host memory...\n");
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    printf("> done\n");
    return 0;
}


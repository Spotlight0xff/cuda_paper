/*
 * Zero-Copy example, using vector addition as showcase
 */
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#define SIZE (1048576)

// CUDA kernel, using zerocopy
__global__ void
vectorAdd(float *A, float *B, float *C, int numElements)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < SIZE) {
        C[id] = A[id] + B[id];
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    float *h_A, *d_A, *h_B, *d_B, *h_C, *d_C;

    // allocate host memory
    h_A = (float*) malloc(SIZE * sizeof(float));
    h_B = (float*) malloc(SIZE * sizeof(float));
    h_C = (float*) malloc(SIZE * sizeof(float));

    // allocate memory on device
    hipMalloc(&d_A, SIZE * sizeof(float));
    hipMalloc(&d_B, SIZE * sizeof(float));
    hipMalloc(&d_C, SIZE * sizeof(float));


    for (int i=0; i < SIZE; i++) {
        h_A[i] = rand() / (float) RAND_MAX;
        h_B[i] = rand() / (float) RAND_MAX;
    }

    hipMemcpy(d_A, h_A, SIZE * sizeof(float), hipMemcpyDefault); // we're using UVA...
    hipMemcpy(d_B, h_B, SIZE * sizeof(float), hipMemcpyDefault);

    printf("> run vectorAdd using copied device memory...\n");
    dim3 block(256);
    dim3 grid((unsigned int) ceil(SIZE / block.x));

    // kernel call
    vectorAdd<<<grid, block>>>(d_A, d_B, d_C, SIZE);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("err: %s\n", hipGetErrorString(err));
    }
    //vectorAdd<<<1024,1024>>>(d_A, d_B, d_C, SIZE);

    hipMemcpy(h_C, d_C, SIZE * sizeof(float), hipMemcpyDefault);

    hipDeviceSynchronize();
    printf("> kernel call synchronized\n");
    printf("%f vs %f\n", h_A[123]+h_B[123], h_C[123]);

    printf("> releasing host memory...\n");

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(h_A);
    hipFree(h_B);
    hipFree(h_C);

    printf("> done\n");
    return 0;
}


/*
 * allocation example - with timing
 * * pinned memory
 * * host memory
 * * device memory
 * * host memory registration (pinned)
 */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../colours.h"


typedef enum {
    PINNED,
    DEVICE,
    HOST_REG
} AllocType;

float profileMemory(AllocType alloc, size_t size, unsigned int flags) {
    hipEvent_t start, stop;
    hipError_t err = hipSuccess;
    void* devPtr = NULL;
    void* hostPtr = malloc(size);
    float ms = 0;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    switch(alloc) {
        case PINNED:
            err = hipHostAlloc(&devPtr, size, flags);
            break;

        case DEVICE:
            err = hipMalloc(&devPtr, size);
            break;

        case HOST_REG:
            err = hipHostRegister(hostPtr, size, flags);
            break;
            
        default:
            fprintf(stderr, " [!] unknown value\n");
    }
    if (err != hipSuccess) {
        fprintf(stderr, "[!] Error: %s\n", hipGetErrorString(err));
        return 0.0f;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    if (devPtr != NULL) {
        hipFree(devPtr);
    }
    if (alloc == HOST_REG) {
        hipHostUnregister(hostPtr);
    }
    if (hostPtr != NULL) {
        free(hostPtr);
    }
    return ms;
}

void printResults(float ms, const char* colour) {
        printf("> %s%f%s ms\n\n", colour, ms, WHITE);
}
/**
 * Host main routine
 */
int
main(void)
{
    size_t samples[] = {512, 1024*1024, 1024*1024*200, 1024*1024*500};
    size_t num_samples = sizeof(samples) / sizeof(samples[0]);
    float ms = 0;
    for(int i=0; i < num_samples; i++) {
        size_t size = samples[i];
        printf("[*] using size %d bytes ( %s%.2f%s MB)\n", size, CYAN, (float)size/1024/1024, WHITE);

        // test device memory
        printf("> profile device memory\n");
        ms = profileMemory(DEVICE, size, 0);
        printResults(ms, RED);

        // test pinned memory
        printf("> profile default pinned memory\n");
        ms = profileMemory(PINNED, size, 0);
        printResults(ms, RED);

        printf("> profile portable pinned memory\n");
        ms = profileMemory(PINNED, size, hipHostMallocPortable);
        printResults(ms, RED);

        printf("> profile mapped pinned memory\n");
        ms = profileMemory(PINNED, size, hipHostMallocMapped);
        printResults(ms, RED);

        printf("> profile wc pinned memory\n");
        ms = profileMemory(PINNED, size, hipHostMallocWriteCombined);
        printResults(ms, RED);

        printf("> profile registered host memory\n");
        ms = profileMemory(HOST_REG, size, hipHostRegisterDefault);
        printResults(ms, RED);
    }
    return 0;
}

